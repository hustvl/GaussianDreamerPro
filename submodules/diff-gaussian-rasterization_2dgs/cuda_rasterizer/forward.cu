#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, bool* clamped)
{
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	clamped[3 * idx + 0] = (result.x < 0);
	clamped[3 * idx + 1] = (result.y < 0);
	clamped[3 * idx + 2] = (result.z < 0);
	return glm::max(result, 0.0f);
}

// Forward version of 2D covariance matrix computation
__device__ float3 computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float* cov3D, const float* viewmatrix)
{
	// The following models the steps outlined by equations 29
	// and 31 in "EWA Splatting" (Zwicker et al., 2002). 
	// Additionally considers aspect / scaling of viewport.
	// Transposes used to account for row-/column-major conventions.
	float3 t = transformPoint4x3(mean, viewmatrix);

	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;

	glm::mat3 J = glm::mat3(
		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0, 0, 0);

	glm::mat3 W = glm::mat3(
		viewmatrix[0], viewmatrix[4], viewmatrix[8],
		viewmatrix[1], viewmatrix[5], viewmatrix[9],
		viewmatrix[2], viewmatrix[6], viewmatrix[10]);

	glm::mat3 T = W * J;

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

	// Apply low-pass filter: every Gaussian should be at least
	// one pixel wide/high. Discard 3rd row and column.
	cov[0][0] += 0.3f;
	cov[1][1] += 0.3f;
	return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
}

// Forward method for converting scale and rotation properties of each
// Gaussian to a 3D covariance matrix in world space. Also takes care
// of quaternion normalization.
__device__ void computeCov3D(const glm::vec3 scale, float mod, const glm::vec4 rot, float* cov3D)
{
	// Create scaling matrix
	glm::mat3 S = glm::mat3(1.0f);
	S[0][0] = mod * scale.x;
	S[1][1] = mod * scale.y;
	S[2][2] = mod * scale.z;

	// Normalize quaternion to get valid rotation
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	// Compute rotation matrix from quaternion
	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::mat3 M = S * R;

	// Compute 3D world covariance matrix Sigma
	glm::mat3 Sigma = glm::transpose(M) * M;

	// Covariance is symmetric, only store upper right
	cov3D[0] = Sigma[0][0];
	cov3D[1] = Sigma[0][1];
	cov3D[2] = Sigma[0][2];
	cov3D[3] = Sigma[1][1];
	cov3D[4] = Sigma[1][2];
	cov3D[5] = Sigma[2][2];
}

// Compute Sxyz in world space for each Gaussian.
__device__ void computeSxyz(const glm::vec3 scale, float mod, const glm::vec4 rot, float* sxyz, const float* viewmatrix)
{
	// Create scaling matrix
	// glm::mat3 S = glm::mat3(1.0f);
	// S[0][0] = mod * scale.x;
	// S[1][1] = mod * scale.y;
	// S[2][2] = mod * scale.z;

	// Normalize quaternion to get valid rotation
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	// Compute rotation matrix from quaternion
	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	// Compute 3D world transformation matrix
	glm::mat3 W = glm::mat3(
		viewmatrix[0], viewmatrix[4], viewmatrix[8],
		viewmatrix[1], viewmatrix[5], viewmatrix[9],
		viewmatrix[2], viewmatrix[6], viewmatrix[10]);

	// Compute final Sxyz matrix: Sxyz = W @ R @ I
	glm::mat3 Sxyz_matrix = R * W;

	// Store Sxyz matrix in row-major order
	sxyz[0] = Sxyz_matrix[0][0];
	sxyz[1] = Sxyz_matrix[1][0];
	sxyz[2] = Sxyz_matrix[2][0];
	sxyz[3] = Sxyz_matrix[0][1];
	sxyz[4] = Sxyz_matrix[1][1];
	sxyz[5] = Sxyz_matrix[2][1];
	sxyz[6] = Sxyz_matrix[0][2];
	sxyz[7] = Sxyz_matrix[1][2];
	sxyz[8] = Sxyz_matrix[2][2];
}

// Perform initial steps for each Gaussian prior to rasterization.
template<int C>
__global__ void preprocessCUDA(int P, int D, int M,
	const float* orig_points,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float tan_fovx, float tan_fovy,
	const float focal_x, float focal_y,
	int* radii,
	float2* points_xy_image,
	float3* means3D_cam,
	float* depths,
	float* cov3Ds,
	float* sxyzs,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	radii[idx] = 0;
	tiles_touched[idx] = 0;

	// Perform near culling, quit if outside.
	float3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;
	means3D_cam[idx] = p_view;

	// Transform point by projecting
	float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
	float4 p_hom = transformPoint4x4(p_orig, projmatrix);
	float p_w = 1.0f / (p_hom.w + 0.0000001f);
	float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };

	// If 3D covariance matrix is precomputed, use it, otherwise compute
	// from scaling and rotation parameters. 
	const float* cov3D;
	if (cov3D_precomp != nullptr)
	{
		cov3D = cov3D_precomp + idx * 6;
	}
	else
	{
		computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
		cov3D = cov3Ds + idx * 6;
	}

	// Compute 2D screen-space covariance matrix
	float3 cov = computeCov2D(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, viewmatrix);

	// Invert covariance (EWA algorithm)
	float det = (cov.x * cov.z - cov.y * cov.y);
	if (det == 0.0f)
		return;
	float det_inv = 1.f / det;
	float3 conic = { cov.z * det_inv, -cov.y * det_inv, cov.x * det_inv };

	// Compute Sxyz in world coordinates
	computeSxyz(scales[idx], scale_modifier, rotations[idx], sxyzs + idx * 9, viewmatrix);

	// Compute extent in screen space (by finding eigenvalues of
	// 2D covariance matrix). Use extent to compute a bounding rectangle
	// of screen-space tiles that this Gaussian overlaps with. Quit if
	// rectangle covers 0 tiles. 
	float mid = 0.5f * (cov.x + cov.z);
	float lambda1 = mid + sqrt(max(0.1f, mid * mid - det));
	float lambda2 = mid - sqrt(max(0.1f, mid * mid - det));
	float my_radius = ceil(3.f * sqrt(max(lambda1, lambda2)));
	float2 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) };
	uint2 rect_min, rect_max;
	getRect(point_image, my_radius, rect_min, rect_max, grid);
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;

	// If colors have been precomputed, use them, otherwise convert
	// spherical harmonics coefficients to RGB color.
	if (colors_precomp == nullptr)
	{
		glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs, clamped);
		rgb[idx * C + 0] = result.x;
		rgb[idx * C + 1] = result.y;
		rgb[idx * C + 2] = result.z;
	}

	// Store some useful helper data for the next steps.
	depths[idx] = p_view.z;
	radii[idx] = my_radius;
	points_xy_image[idx] = point_image;
	// Inverse 2D covariance and opacity neatly pack into one float4
	conic_opacity[idx] = { conic.x, conic.y, conic.z, opacities[idx] };
	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);
}

// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching 
// and rasterizing data.
template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float focal_x, const float focal_y,
	const bool use_integral,
	const float scale_modifier,
	const float2* __restrict__ points_xy_image,
	const float* __restrict__ features,
	const float4* __restrict__ conic_opacity,
	const float* __restrict__ sxyz,
	const float3* __restrict__ scales,
	const float3* __restrict__ means3D_cam,
	float* __restrict__ final_T,
	uint32_t* __restrict__ n_contrib,
	const float* __restrict__ bg_color,
	float* __restrict__ out_color,
	float* __restrict__ out_depth,
	float* __restrict__ out_normal,
	float* __restrict__ out_accum)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y };

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];
	__shared__ float3 collected_sx[BLOCK_SIZE];
	__shared__ float3 collected_sy[BLOCK_SIZE];
	__shared__ float3 collected_sz[BLOCK_SIZE];
	__shared__ float3 collected_means3D_cam[BLOCK_SIZE];
	__shared__ float3 collected_scales[BLOCK_SIZE];

	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	float C[CHANNELS] = { 0 };
	float Depth = 0.0f;
	float preDepth = 0.0f;
	float Normal[3] = { 0 };
	// for ray rotation
	float W_center = 0.5f * W - 0.5f;
	float H_center = 0.5f * H - 0.5f;
	float focal_x_inv = 1.0f/focal_x;
	float focal_y_inv = 1.0f/focal_y;
	float3 view_dir = { (pixf.x-W_center)*focal_x_inv, (pixf.y-H_center)*focal_y_inv, 1.0f };
	float dep2dist = sqrt(view_dir.x*view_dir.x + view_dir.y*view_dir.y + 1.0f);
	float dist2dep = 1.0f / dep2dist;
	view_dir = { view_dir.x * dist2dep, view_dir.y * dist2dep, view_dir.z * dist2dep };

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
			int coll_id9 = coll_id * 9;
			collected_sx[block.thread_rank()] = { sxyz[coll_id9 + 0], sxyz[coll_id9 + 1], sxyz[coll_id9 + 2] };
			collected_sy[block.thread_rank()] = { sxyz[coll_id9 + 3], sxyz[coll_id9 + 4], sxyz[coll_id9 + 5] };
			collected_sz[block.thread_rank()] = { sxyz[coll_id9 + 6], sxyz[coll_id9 + 7], sxyz[coll_id9 + 8] };
			collected_means3D_cam[block.thread_rank()] = means3D_cam[coll_id];
			collected_scales[block.thread_rank()] = scales[coll_id];
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			contributor++;

			// Compute hit point in camera space and gaussian space
			float3 sx = collected_sx[j];
			float3 sy = collected_sy[j];
			float3 sz = collected_sz[j];
			float3 xyz_cam = collected_means3D_cam[j];
			float3 scale = collected_scales[j];
			scale = { scale.x * scale_modifier, scale.y * scale_modifier, scale.z * scale_modifier }; // TODO: do this in process
			float n_dot_c = sz.x * xyz_cam.x + sz.y * xyz_cam.y + sz.z * xyz_cam.z; // TODO: do this in process
			float n_dot_view = sz.x * view_dir.x + sz.y * view_dir.y + sz.z * view_dir.z;
			// if the gaussian plane is nearly parallel to the ray, skip
			if (abs(n_dot_view) < 1e-9)
				continue;
			const float n_dot_view_inv = 1.0f / n_dot_view;
			float distance = n_dot_c * n_dot_view_inv;
			float3 hit_pt = { distance * view_dir.x, distance * view_dir.y, distance * view_dir.z };
			float3 offset_cam = { hit_pt.x - xyz_cam.x, hit_pt.y - xyz_cam.y, hit_pt.z - xyz_cam.z };
			float2 uv = { sx.x * offset_cam.x + sx.y * offset_cam.y + sx.z * offset_cam.z, sy.x * offset_cam.x + sy.y * offset_cam.y + sy.z * offset_cam.z };
			// 3 sigma
			float2 scale_inv = { 1.0f / scale.x, 1.0f / scale.y };
			const float x_div_sx = uv.x * scale_inv.x;
			const float y_div_sy = uv.y * scale_inv.y;
			if ((abs(x_div_sx) > 3.0f) || (abs(y_div_sy) > 3.0f))
				continue;

			float G = 0.0f;
			bool screen_filtering = false;
			if (!use_integral){
				float power = -0.5f * (x_div_sx * x_div_sx + y_div_sy * y_div_sy);
				G = exp(power);
				// // screen filtering
				// float2 xy = collected_xy[j];
				// float2 d = { xy.x - pixf.x, xy.y - pixf.y };
				// float power_screen = -(d.x * d.x + d.y * d.y);
				// if (power > power_screen)
				// 	G = exp(power);
				// else {
				// 	G = exp(power_screen);
				// 	screen_filtering = true;
				// }
			}
			else {
				float radius = 0.25f * hit_pt.z * (focal_x_inv + focal_y_inv) * abs(n_dot_view_inv);
				const float r_div_sx = radius * scale_inv.x;
				const float r_div_sy = radius * scale_inv.y;
				const float z_inv2 = 1.0f / (hit_pt.z * hit_pt.z);
				G = _2pi * abs(n_dot_view) * scale.x * scale.y * focal_x * focal_y * z_inv2
					* (F_cdf(x_div_sx + r_div_sx) - F_cdf(x_div_sx - r_div_sx))
					* (F_cdf(y_div_sy + r_div_sy) - F_cdf(y_div_sy - r_div_sy));
			}
			
			// Resample using conic matrix (cf. "Surface 
			// Splatting" by Zwicker et al., 2001)
			// float2 xy = collected_xy[j];
			// float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			float4 con_o = collected_conic_opacity[j];
			// float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;

			// Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix). 
			float alpha = min(0.99f, con_o.w * G);
			if (alpha < 1.0f / 255.0f)
				continue;
			float test_T = T * (1 - alpha);
			if (test_T < 0.0001f)
			{
				done = true;
				continue;
			}

			// Eq. (3) from 3D Gaussian splatting paper.
			float weight = alpha * T;
			for (int ch = 0; ch < CHANNELS; ch++)
				C[ch] += features[collected_id[j] * CHANNELS + ch] * weight;
			Depth += distance * weight; // weighted depth
			// midpoint depth
			// if (preDepth == 0.0f)
			// 	preDepth = distance;
			// if (T > 0.5f && test_T <= 0.5f)
			// 	Depth = preDepth + (distance - preDepth) * (T - 0.5f) / (T - test_T);
			// preDepth = distance;
			float normal_sign = n_dot_view < 0.0f ? 1.0f : -1.0f;
			float signed_weight = normal_sign * weight;
			Normal[0] = Normal[0] + signed_weight * sz.x;
			Normal[1] = Normal[1] + signed_weight * sz.y;
			Normal[2] = Normal[2] + signed_weight * sz.z;

			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			last_contributor = contributor;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;
		for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
		// TODO: normalize according to final_T?
		out_depth[pix_id] = Depth;
		for (int ch = 0; ch < 3; ch++)
			out_normal[ch * H * W + pix_id] = Normal[ch];
		out_accum[pix_id] = 1.0f - T;
	}
}

void FORWARD::render(
	const dim3 grid, dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const float focal_x, const float focal_y,
	const bool use_integral,
	const float scale_modifier,
	const float2* means2D,
	const float* colors,
	const float4* conic_opacity,
	const float* sxyz,
	const float3* scales,
	const float3* means3D_cam,
	float* final_T,
	uint32_t* n_contrib,
	const float* bg_color,
	float* out_color,
	float* out_depth,
	float* out_normal,
	float* out_accum)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> > (
		ranges,
		point_list,
		W, H,
		focal_x, focal_y,
		use_integral,
		scale_modifier,
		means2D,
		colors,
		conic_opacity,
		sxyz,
		scales,
		means3D_cam,
		final_T,
		n_contrib,
		bg_color,
		out_color,
		out_depth,
		out_normal,
		out_accum);
}

void FORWARD::preprocess(int P, int D, int M,
	const float* means3D,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	int* radii,
	float2* means2D,
	float3* means3D_cam,
	float* depths,
	float* cov3Ds,
	float* sxyz,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	preprocessCUDA<NUM_CHANNELS> << <(P + 255) / 256, 256 >> > (
		P, D, M,
		means3D,
		scales,
		scale_modifier,
		rotations,
		opacities,
		shs,
		clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, 
		projmatrix,
		cam_pos,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		radii,
		means2D,
		means3D_cam,
		depths,
		cov3Ds,
		sxyz,
		rgb,
		conic_opacity,
		grid,
		tiles_touched,
		prefiltered
		);
}